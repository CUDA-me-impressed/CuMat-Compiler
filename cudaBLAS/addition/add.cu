/*
    This file consists of CUDA code which is compiled with the CuMat
    program and linked in with the output with clang.

    Most of this is just setup for llvm with cuBLAS so that we can
    call the functions and have them return CuMat data (row major)
*/
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>


void CuMatAddMatrixD(double * matA, double * matB, double * matRes, long i, long j){
    // Pointers for the various kernel vars
    double *a, *b, *res;

    // Length calculations
    long N = i*j;
    size_t matSize = N*sizeof(double);

    // Allocate on device
    hipMallocManaged(&a, matSize);
    hipMallocManaged(&b, matSize);
    hipMallocManaged(&res, matSize);

    // Copy from host to device
    hipblasSetVector(N, sizeof(double), matA, 1, a, 1);
    hipblasSetVector(N, sizeof(double), matB, 1, b, 1);

    // Create cublas handler
    hipblasHandle_t h;
    hipblasCreate(&h);

    // Carry out addition
    const double scale = 1;
    hipblasDaxpy(h, N, &scale, a,1, b, 1);
    // Copy vector off gpu
    hipblasGetVector(N, sizeof(double), b, 1, res, 1);

    // Copy the results out of device memory
    hipMemcpy(matRes, res, matSize, hipMemcpyDeviceToHost);

    // Destroy cublas
    hipblasDestroy(h);

    // Free device memory
    hipFree(a);
    hipFree(b);
}

// Device function
__global__ void CuMatAddMatrixIKernel(long* A, long* B, long * res, long i, long j){
    long N = i * j; // Treat matrix add as vector add (same thing for equal sizes)
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < N){
        res[index] = A[index] + B[index];
    }
}


void CuMatAddMatrixI(long * matA, long * matB, long * matRes, long i, long j){
    long* d_A; long *d_B; long * d_Res;
    size_t size = i*j*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = ((i*j) + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatAddMatrixIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, i, j);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}