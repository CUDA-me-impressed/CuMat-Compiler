/*
    This file consists of CUDA code which is compiled with the CuMat
    program and linked in with the output with clang.

    Most of this is just setup for llvm with cuBLAS so that we can
    call the functions and have them return CuMat data (row major)
*/
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>


void CuMatAddMatrixD(double * matA, double * matB, double * matRes, long i, long j){
    // Pointers for the various kernel vars
    double *a, *b, *res;

    // Length calculations
    long N = i*j;
    size_t matSize = N*sizeof(double);

    // Res needs to be allocated on the host
    res = (double*) malloc(matSize);
    // Allocate on device
    hipMallocManaged(&a, matSize);
    hipMallocManaged(&b, matSize);

    // Copy from host to device
    hipblasSetVector(N, sizeof(double), matA, 1, a, 1);
    hipblasSetVector(N, sizeof(double), matB, 1, b, 1);

    // Create cublas handler
    hipblasHandle_t h;
    hipblasCreate(&h);

    // Carry out addition
    const double scale = 1;
    hipblasDaxpy(h, N, &scale, a,1, b, 1);
    // Copy vector off gpu
    hipblasGetVector(N, sizeof(double), b, 1, res, 1);

    // Destroy cublas
    hipblasDestroy(h);
}