/*
    This file consists of CUDA code which is compiled with the CuMat
    program and linked in with the output with clang.

    Most of this is just setup for llvm with cuBLAS so that we can
    call the functions and have them return CuMat data (row major)
*/
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>


#define BLOCK_SIZE 32 // nvidia GPUs typically have 1024 threads per block, 32*32

__global__ void CuMatMatMultKernelI(const long *matA, const long *matB, long* matRes, int width, int i, int j)
{
    // Get out the indicies for the multiplication
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    // check boundry conditions incase block size is at the end
    if( r < i && c < j){
        long value = 0;
        for(int k = 0; k < width; k++){
            value += matA[r * width + k] * matB[k * j + c];
        }
        matRes[r * j + c] = value;
    }
}

void CuMatMatMultMatrixI(long * matA, long * matB, long * matRes, long i, long p, long j){
    auto matASize = sizeof(long) * i * p;
    auto matBSize = sizeof(long) * p * j;
    auto matResSize = i * j * sizeof(long);

    hipStream_t stream;

    // Allocate device memory
    long *d_A, *d_B, *d_Res;

    if (matRes == NULL) {
        exit(EXIT_FAILURE);
    }

    hipMallocManaged(&d_A, matASize);
    hipMallocManaged(&d_B, matBSize);
    hipMallocManaged(&d_Res, matResSize);

    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    // copy host memory to device
    hipMemcpy(d_A, matA, matASize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, matBSize, hipMemcpyHostToDevice);

    // Setup execution parameters
    dim3 dim_grid(ceilf(i/(float)BLOCK_SIZE), ceilf(j/(float)BLOCK_SIZE), 1);
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);


    CuMatMatMultKernelI<<<dim_grid, dim_block>>>(d_A, d_B, d_Res, p,  i, j);

    // Copy result from device to host
    hipMemcpyAsync(matRes, d_Res, matResSize, hipMemcpyDeviceToHost, stream);
    // Copy the results out of device memory
//    cudaMemcpy(matRes, d_Res, matResSize, cudaMemcpyDeviceToHost);
    hipStreamSynchronize(stream);

    // Clean up memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_Res);
}

// matRes(m,n) = matA(m,k) * matB(k,n)
void CuMatMatMultMatrixD(const double *matA, const double *matB, double *matRes, const int m, const int k, const int n) {
    // Declare matA, matB, matRes on device
    double* d_A;
    double* d_B;
    double* d_Res;

    size_t matASize = m * k * sizeof(double);
    size_t matBSize = k * n * sizeof(double);
    size_t matResSize = m * n * sizeof(double);

    // Allocate memory for device
    hipMalloc(&d_A, matASize);
    hipMalloc(&d_B, matBSize);
    hipMalloc(&d_Res, matResSize);

    // Copy over matA & matB to device
    hipMemcpy(d_A, matA, matASize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, matASize, hipMemcpyHostToDevice);

    int lda=m,ldb=k,ldc=m;
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const double alpha = 1.0f;
    const double beta = 0.0f;

    // Do the actual multiplication
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_B, lda, d_A, ldb, &beta, d_Res, ldc);

    // Synchronise before copy
    hipDeviceSynchronize();

    // Copy device memory to host
    hipMemcpy(matRes,d_Res,matResSize,hipMemcpyDeviceToHost);

    // Destroy the handle
    hipblasDestroy(handle);
}


/*
 * Multiplication for element wise operations
 */

// Device function
__global__ void CuMatElementMultMatrixDKernel(double* A, double* B, double * res, long i, long j){
    long N = i * j; // Treat matrix add as vector add (same thing for equal sizes)
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < N){
        res[index] = A[index] * B[index];
    }
}

void CuMatElementMultMatrixD(double * matA, double * matB, double * matRes, long i, long j){
    double* d_A; double *d_B; double * d_Res;
    size_t size = i*j*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = ((i*j) + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatElementMultMatrixDKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, i, j);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}

// Device function
__global__ void CuMatElementMultMatrixKernelI(long* A, long* B, long * res, long i, long j){
    long N = i * j; // Treat matrix add as vector add (same thing for equal sizes)
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < N){
        res[index] = A[index] * B[index];
    }
}


void CuMatElementMultMatrixI(long * matA, long * matB, long * matRes, long i, long j){
    long* d_A; long *d_B; long * d_Res;
    size_t size = i*j*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = ((i*j) + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatElementMultMatrixKernelI<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, i, j);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}