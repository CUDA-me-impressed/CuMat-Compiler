#include "hip/hip_runtime.h"
/*
    This file consists of CUDA code which is compiled with the CuMat
    program and linked in with the output with clang.

    Most of this is just setup for llvm with cuBLAS so that we can
    call the functions and have them return CuMat data (row major)
*/
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../utils/headers.hpp"
#define BLOCK_SIZE 32 // nvidia GPUs typically have 1024 threads per block, 32*32

__global__ void CuMatMatMultKernelI(const long *matA, const long *matB, long* matRes, int width, int i, int j)
{
    // Get out the indicies for the multiplication
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    // check boundry conditions incase block size is at the end
    if( r < i && c < j){
        long value = 0;
        for(int k = 0; k < width; k++){
            value += matA[r * width + k] * matB[k * j + c];
        }
        matRes[r * j + c] = value;
    }
}

extern "C" void CuMatMatMultMatrixI(HeaderI* matHeaderA, HeaderI* matHeaderB, HeaderI* matHeaderRes, long i, long p, long j){
    long* matA;
    long* matB;
    long* matRes;
    matA = matHeaderA->data;
    matB = matHeaderB->data;
    matRes = matHeaderRes->data;
    auto matASize = sizeof(long) * i * p;
    auto matBSize = sizeof(long) * p * j;
    auto matResSize = i * j * sizeof(long);

    hipStream_t stream;

    // Allocate device memory
    long *d_A, *d_B, *d_Res;

    if (matRes == NULL) {
        exit(EXIT_FAILURE);
    }

    hipMallocManaged(&d_A, matASize);
    hipMallocManaged(&d_B, matBSize);
    hipMallocManaged(&d_Res, matResSize);

    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    // copy host memory to device
    hipMemcpy(d_A, matA, matASize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, matBSize, hipMemcpyHostToDevice);

    // Setup execution parameters
    dim3 dim_grid(ceilf(i/(float)BLOCK_SIZE), ceilf(j/(float)BLOCK_SIZE), 1);
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);


    CuMatMatMultKernelI<<<dim_grid, dim_block>>>(d_A, d_B, d_Res, p,  i, j);

    // Copy result from device to host
    hipMemcpyAsync(matRes, d_Res, matResSize, hipMemcpyDeviceToHost, stream);
    // Copy the results out of device memory
//    hipMemcpy(matRes, d_Res, matResSize, hipMemcpyDeviceToHost);
    hipStreamSynchronize(stream);

    // Clean up memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_Res);
}

// matRes(m,n) = matA(m,k) * matB(k,n)
extern "C" void CuMatMatMultMatrixD(HeaderD* matHeaderA, HeaderD* matHeaderB, HeaderD* matHeaderRes, const int m, const int k, const int n) {
    double* matA;
    double* matB;
    double* matRes;
    matA = matHeaderA->data;
    matB = matHeaderB->data;
    matRes = matHeaderRes->data;
    // Declare matA, matB, matRes on device
    double* d_A;
    double* d_B;
    double* d_Res;

    size_t matASize = m * k * sizeof(double);
    size_t matBSize = k * n * sizeof(double);
    size_t matResSize = m * n * sizeof(double);

    // Allocate memory for device
    hipMallocManaged(&d_A, matASize);
    hipMallocManaged(&d_B, matBSize);
    hipMallocManaged(&d_Res, matResSize);

    // Copy over matA & matB to device
    hipMemcpy(d_A, matA, matASize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, matASize, hipMemcpyHostToDevice);

    int lda=m,ldb=k,ldc=m;
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const double alpha = 1.0f;
    const double beta = 0.0f;

    // Do the actual multiplication
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_B, lda, d_A, ldb, &beta, d_Res, ldc);

    // Synchronise before copy
    hipDeviceSynchronize();

    // Copy device memory to host
    hipMemcpy(matRes,d_Res,matResSize,hipMemcpyDeviceToHost);

    // Destroy the handle
    hipblasDestroy(handle);
}