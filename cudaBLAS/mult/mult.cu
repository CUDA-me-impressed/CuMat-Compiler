/*
    This file consists of CUDA code which is compiled with the CuMat
    program and linked in with the output with clang.

    Most of this is just setup for llvm with cuBLAS so that we can
    call the functions and have them return CuMat data (row major)
*/
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void CuMatMultMatrixI(long * matA, long * matB, long * matRes, long i, long p, long j){
    auto matASize = sizeof(long) * i * p;
    auto matBSize = sizeof(long) * p * j;
    auto matResSize = i * j * sizeof(long);

    long* h_A; long * h_B; long *h_Res;

    // Allocate cuda managed host memory
    hipMallocManaged(&h_A, matASize);
    hipMallocManaged(&h_B, matBSize);
    hipMallocManaged(&h_Res, matResSize);

    // Copy over the data from the function pointers
    hipMemcpy(h_A, matA, matASize, hipMemcpyHostToDevice);
    hipMemcpy(h_B, matB, matBSize, hipMemcpyHostToDevice);

    hipStream_t stream;

    // Allocate device memory
    float *d_A, *d_B, *d_Res;


    // Allocate host matrix C
    float *h_C;
    hipHostMalloc(&h_Res, matResSize);

    if (h_C == NULL) {
        exit(EXIT_FAILURE);
    }

    hipMalloc(reinterpret_cast<void **>(&d_A), matASize);
    hipMalloc(reinterpret_cast<void **>(&d_B), matBSize);
    hipMalloc(reinterpret_cast<void **>(&d_C), matResSize);

    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    // copy host memory to device
    hipMemcpyAsync(d_A, h_A, matASize, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, h_B, matBSize, hipMemcpyHostToDevice, stream);

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(p / threads.x, p / threads.y);

    // Performs warmup operation using matrixMul CUDA kernel
    if (block_size == 16) {
        MatrixMulCUDA<16>
            <<<grid, threads, 0, stream>>>(d_Res, d_A, d_B, i, p);
    } else {
        MatrixMulCUDA<32>
            <<<grid, threads, 0, stream>>>(d_Res, d_A, d_B, i, p);
    }

    hipStreamSynchronize(stream);

    // Execute the kernel
    int nIter = 300;

    for (int itt = 0; itt < nIter; itt++) {
        if (block_size == 16) {
        MatrixMulCUDA<16>
            <<<grid, threads, 0, stream>>>(d_Res, d_A, d_B, i, p);
        } else {
        MatrixMulCUDA<32>
            <<<grid, threads, 0, stream>>>(d_Res, d_A, d_B, i, p);
        }
    }

    // Copy result from device to host
    hipMemcpyAsync(h_Res, d_Res, matResSize, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    // Clean up memory
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}