#include "hip/hip_runtime.h"
/*
 * Generalised Matrix Vector multiplication cuda functions
 */

#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../utils/headers.hpp"

__global__ void CuMatGEMVKernelI(long* vec, long* mat, long* res, long N, long M) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    long sum = 0;
    if (tid < M) {
        for (int i = 0; i < N; i++) sum += vec[i] * mat[(i * M) + tid];
        res[tid] = sum;
    }
}

extern "C" void CuMatGEMVMatrixI(HeaderI* matHeaderA, HeaderI* matHeaderB, HeaderI* matHeaderRes, long N, long M) {
    long* vecA;
    long* matB;
    long* vecRes;

    vecA = matHeaderA->data;
    matB = matHeaderB->data;
    vecRes = matHeaderRes->data;

    long *d_A, *d_B, *d_Res;

    hipMalloc((void**)&d_A, sizeof(long) * N);
    hipMalloc((void**)&d_B, sizeof(long) * N * M);
    hipMalloc((void**)&d_Res, sizeof(long) * M);

    hipMemcpy(d_A, vecA, sizeof(long) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, sizeof(long) * N * M, hipMemcpyHostToDevice);

    // Takes in Vector, Matrix, Resultant Vector and sizes
    CuMatGEMVKernelI<<<M / 256 + 1, 256>>>(d_A, d_B, d_Res, N, M);

    hipMemcpy(vecRes, d_Res, sizeof(long) * M, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_Res);
}