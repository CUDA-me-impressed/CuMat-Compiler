/*
    This file consists of CUDA code which is compiled with the CuMat
    program and linked in with the output with clang.

    Most of this is just setup for llvm with cuBLAS so that we can
    call the functions and have them return CuMat data (row major)
*/
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>


// Device function
__global__ void CuMatLORMatrixDKernel(double* A, double* B, double * res, long i, long j){
    long N = i * j; // Treat matrix add as vector add (same thing for equal sizes)
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < N){
        res[index] = (double) (A[index] || B[index]);
    }
}

void CuMatLORMatrixD(double * matA, double * matB, double * matRes, long i, long j){
    double* d_A; double *d_B; double * d_Res;
    size_t size = i*j*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = ((i*j) + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatLORMatrixDKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, i, j);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}

// Device function
__global__ void CuMatLORMatrixIKernel(long* A, long* B, long * res, long i, long j){
    long N = i * j; // Treat matrix add as vector add (same thing for equal sizes)
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < N){
        res[index] = (long) (A[index] || B[index]);
    }
}


void CuMatLORMatrixI(long * matA, long * matB, long * matRes, long i, long j){
    long* d_A; long *d_B; long * d_Res;
    size_t size = i*j*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = ((i*j) + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatLORMatrixIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, i, j);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}


/*
 * Logical AND Kernel functions
 */


// Device function
__global__ void CuMatLANDMatrixDKernel(double* A, double* B, double * res, long i, long j){
    long N = i * j; // Treat matrix add as vector add (same thing for equal sizes)
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < N){
        res[index] = (double) (A[index] && B[index]);
    }
}

void CuMatLANDMatrixD(double * matA, double * matB, double * matRes, long i, long j){
    double* d_A; double *d_B; double * d_Res;
    size_t size = i*j*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = ((i*j) + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatLANDMatrixDKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, i, j);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}

// Device function
__global__ void CuMatLANDMatrixIKernel(long* A, long* B, long * res, long i, long j){
    long N = i * j; // Treat matrix add as vector add (same thing for equal sizes)
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < N){
        res[index] = (long) (A[index] && B[index]);
    }
}


void CuMatLANDMatrixI(long * matA, long * matB, long * matRes, long i, long j){
    long* d_A; long *d_B; long * d_Res;
    size_t size = i*j*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = ((i*j) + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatLANDMatrixIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, i, j);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}