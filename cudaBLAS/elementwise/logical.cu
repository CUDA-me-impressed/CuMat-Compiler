/*
    This file consists of CUDA code which is compiled with the CuMat
    program and linked in with the output with clang.

    Most of this is just setup for llvm with cuBLAS so that we can
    call the functions and have them return CuMat data (row major)
*/
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>


// Device function
__global__ void CuMatLORMatrixDKernel(double* A, double* B, double * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = (double) (A[index] || B[index]);
    }
}

extern "C" void CuMatLORMatrixD(double * matA, double * matB, double * matRes, long len){
    double* d_A; double *d_B; double * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatLORMatrixDKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}

// Device function
__global__ void CuMatLORMatrixIKernel(long* A, long* B, long * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = (long) (A[index] || B[index]);
    }
}


extern "C" void CuMatLORMatrixI(long * matA, long * matB, long * matRes, long len){
    long* d_A; long *d_B; long * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatLORMatrixIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}


/*
 * Logical AND Kernel functions
 */


// Device function
__global__ void CuMatLANDMatrixDKernel(double* A, double* B, double * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = (double) (A[index] && B[index]);
    }
}

extern "C" void CuMatLANDMatrixD(double * matA, double * matB, double * matRes, long len){
    double* d_A; double *d_B; double * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatLANDMatrixDKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}

// Device function
__global__ void CuMatLANDMatrixIKernel(long* A, long* B, long * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = (long) (A[index] && B[index]);
    }
}


void CuMatLANDMatrixI(long * matA, long * matB, long * matRes, long len){
    long* d_A; long *d_B; long * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatLANDMatrixIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}
