/*
    This file consists of CUDA code which is compiled with the CuMat
    program and linked in with the output with clang.

    Most of this is just setup for llvm with cuBLAS so that we can
    call the functions and have them return CuMat data (row major)
*/
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>


// Device function
__global__ void CuMatLTMatrixDKernel(double* A, double* B, double * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = static_cast<double>(A[index] < B[index]);
    }
}

extern "C" void CuMatLTMatrixD(double * matA, double * matB, double * matRes, long len){
    double* d_A; double *d_B; double * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatLTMatrixDKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}

// Device function
__global__ void CuMatLTMatrixIKernel(long* A, long* B, long * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = static_cast<long>(A[index] < B[index]);
    }
}


extern "C" void CuMatLTMatrixI(long * matA, long * matB, long * matRes, long len){
    long* d_A; long *d_B; long * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatLTMatrixIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}




// Device function
__global__ void CuMatGTMatrixDKernel(double* A, double* B, double * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = static_cast<double>(A[index] > B[index]);
    }
}

extern "C" void CuMatGTMatrixD(double * matA, double * matB, double * matRes, long len){
    double* d_A; double *d_B; double * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatGTMatrixDKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}

// Device function
__global__ void CuMatGTMatrixIKernel(long* A, long* B, long * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = static_cast<long>(A[index] > B[index]);
    }
}


extern "C" void CuMatGTMatrixI(long * matA, long * matB, long * matRes, long len){
    long* d_A; long *d_B; long * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatGTMatrixIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}




// Device function
__global__ void CuMatLTEMatrixDKernel(double* A, double* B, double * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = static_cast<double>(A[index] <= B[index]);
    }
}

extern "C" void CuMatLTEMatrixD(double * matA, double * matB, double * matRes, long len){
    double* d_A; double *d_B; double * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatLTEMatrixDKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}

// Device function
__global__ void CuMatLTEMatrixIKernel(long* A, long* B, long * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = static_cast<long>(A[index] <= B[index]);
    }
}


extern "C" void CuMatLTEMatrixI(long * matA, long * matB, long * matRes, long len){
    long* d_A; long *d_B; long * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatLTEMatrixIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}



// Device function
__global__ void CuMatGTEMatrixDKernel(double* A, double* B, double * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = static_cast<double>(A[index] >= B[index]);
    }
}

extern "C" void CuMatGTEMatrixD(double * matA, double * matB, double * matRes, long len){
    double* d_A; double *d_B; double * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatGTEMatrixDKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}

// Device function
__global__ void CuMatGTEMatrixIKernel(long* A, long* B, long * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = static_cast<long>(A[index] >= B[index]);
    }
}


extern "C" void CuMatGTEMatrixI(long * matA, long * matB, long * matRes, long len){
    long* d_A; long *d_B; long * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatGTEMatrixIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}




// Device function
__global__ void CuMatEQMatrixDKernel(double* A, double* B, double * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = static_cast<double>(A[index] == B[index]);
    }
}

extern "C" void CuMatEQMatrixD(double * matA, double * matB, double * matRes, long len){
    double* d_A; double *d_B; double * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatEQMatrixDKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}

// Device function
__global__ void CuMatEQMatrixIKernel(long* A, long* B, long * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = static_cast<long>(A[index] == B[index]);
    }
}


extern "C" void CuMatEQMatrixI(long * matA, long * matB, long * matRes, long len){
    long* d_A; long *d_B; long * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatEQMatrixIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}




// Device function
__global__ void CuMatNEQMatrixDKernel(double* A, double* B, double * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = static_cast<double>(A[index] != B[index]);
    }
}

extern "C" void CuMatNEQMatrixD(double * matA, double * matB, double * matRes, long len){
    double* d_A; double *d_B; double * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatNEQMatrixDKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}

// Device function
__global__ void CuMatNEQMatrixIKernel(long* A, long* B, long * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = static_cast<long>(A[index] != B[index]);
    }
}


extern "C" void CuMatNEQMatrixI(long * matA, long * matB, long * matRes, long len){
    long* d_A; long *d_B; long * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatNEQMatrixIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}