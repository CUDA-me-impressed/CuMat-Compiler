/*
    This file consists of CUDA code which is compiled with the CuMat
    program and linked in with the output with clang.

    Most of this is just setup for llvm with cuBLAS so that we can
    call the functions and have them return CuMat data (row major)
*/
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

extern "C" void CuMatAddMatrixD(double * matA, double * matB, double * matRes, long len){
    // Pointers for the various kernel vars
    double *a, *b, *res;

    // Length calculations
    size_t matSize = len*sizeof(double);

    // Allocate on device
    hipMallocManaged(&a, matSize);
    hipMallocManaged(&b, matSize);
    hipMallocManaged(&res, matSize);

    // Copy from host to device
    hipblasSetVector(len, sizeof(double), matA, 1, a, 1);
    hipblasSetVector(len, sizeof(double), matB, 1, b, 1);

    // Create cublas handler
    hipblasHandle_t h;
    hipblasCreate(&h);

    // Carry out addition
    const double scale = 1;
    hipblasDaxpy(h, len, &scale, a,1, b, 1);
    // Copy vector off gpu
    hipblasGetVector(len, sizeof(double), b, 1, res, 1);

    // Copy the results out of device memory
    hipMemcpy(matRes, res, matSize, hipMemcpyDeviceToHost);

    // Destroy cublas
    hipblasDestroy(h);

    // Free device memory
    hipFree(a);
    hipFree(b);
}

// Device function
__global__ void CuMatAddMatrixIKernel(long* A, long* B, long * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = A[index] + B[index];
    }
}


extern "C" void CuMatAddMatrixI(long* matA, long* matB, long* matRes, long len){
    long* d_A; long *d_B; long * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = ((len) + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatAddMatrixIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    auto result = hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    if(result != hipSuccess){
        printf("GPUassert: %s\n", hipGetErrorString(result));
    }

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}
