#include "hip/hip_runtime.h"
/*
    This file consists of CUDA code which is compiled with the CuMat
    program and linked in with the output with clang.

    Most of this is just setup for llvm with cuBLAS so that we can
    call the functions and have them return CuMat data (row major)
*/
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../utils/headers.hpp"

// Device function
__global__ void CuMatDivMatrixDKernel(double* A, double* B, double * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = static_cast<double>(A[index] / B[index]);
    }
}

extern "C" void CuMatDivMatrixD(HeaderD* matHeaderA, HeaderD* matHeaderB, HeaderD* matHeaderRes, long len){
    double* matA;
    double* matB;
    double* matRes;
    matA = matHeaderA->data;
    matB = matHeaderB->data;
    matRes = matHeaderRes->data;
    double* d_A; double *d_B; double * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatDivMatrixDKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}

// Device function
__global__ void CuMatDivMatrixIKernel(long* A, long* B, long * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = static_cast<long>(A[index] / B[index]);
    }
}


extern "C" void CuMatDivMatrixI(HeaderI* matHeaderA, HeaderI* matHeaderB, HeaderI* matHeaderRes, long len){
    long* matA;
    long* matB;
    long* matRes;
    matA = matHeaderA->data;
    matB = matHeaderB->data;
    matRes = matHeaderRes->data;
    long* d_A; long *d_B; long * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatDivMatrixIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}