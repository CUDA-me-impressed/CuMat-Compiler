/*
    This file consists of CUDA code which is compiled with the CuMat
    program and linked in with the output with clang.

    Most of this is just setup for llvm with cuBLAS so that we can
    call the functions and have them return CuMat data (row major)
*/
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>


// Device function
__global__ void CuMatSubMatrixDKernel(double* A, double* B, double * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = A[index] - B[index];
    }
}

extern "C" void CuMatSubMatrixD(double * matA, double * matB, double * matRes, long len){
    double* d_A; double *d_B; double * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatSubMatrixDKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}

// Device function
__global__ void CuMatSubMatrixIKernel(long* A, long* B, long * res, long len){
    long index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < len){
        res[index] = A[index] - B[index];
    }
}


extern "C" void CuMatSubMatrixI(long * matA, long * matB, long * matRes, long len){
    long* d_A; long *d_B; long * d_Res;
    size_t size = len*sizeof(long);
    // Allocate memory for CUDA
    hipMallocManaged(&d_A, size);
    hipMallocManaged(&d_B, size);
    hipMallocManaged(&d_Res, size);

    // Copy over the matricies into device memory
    hipMemcpy(d_A, matA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matB, size, hipMemcpyHostToDevice);

    // Set the number of threads per block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock -1) / threadsPerBlock;

    // Call the kernel
    CuMatSubMatrixIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_Res, len);

    // Synchronise before copying
    hipDeviceSynchronize();

    // Copy the results out of device memory
    hipMemcpy(matRes, d_Res, size, hipMemcpyDeviceToHost);

    // Free up cuda malloc
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(&d_Res);
}
