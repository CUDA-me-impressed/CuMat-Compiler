#include "elementwise/add.cu"
#include <iostream>

int main(){
    long A[6] = {1,2,7,3,4,1};
    long B[6] = {4,3,8,2,1,3};
    long C[6] = {0,0,0,0,0,0};

    CuMatAddMatrixI(A,B,C,2,3);
    for(int i = 0; i < 6; i++){
        std::cout << C[i] << std::endl;
    }
}
